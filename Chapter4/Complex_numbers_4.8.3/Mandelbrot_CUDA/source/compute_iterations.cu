#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


__global__ void kernel(int* iter,float xdim,float ydim,int max_iter)
{
	
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	float x = (float)blockIdx.x;
	float y = (float)threadIdx.x;
	x *= 2.4f / ydim;
	y *= 2.4f / ydim;
	x -= 1.2 * (xdim / ydim) + 0.5;
	y -= 1.2;
	float z_x = x;
	float z_y = y;
	float temp;
	for (int j = 0; j < max_iter; j++) {
		if ((z_x * z_x + z_y * z_y) > 4.0f) { break; }
		iter[i]++;
		temp = z_x;
		z_x = z_x * z_x - z_y * z_y + x;
		z_y = 2 * z_y * temp + y;
	}
}


void compute_iterations(int* iter,int xdim,int ydim, int max_iter) {

	int N = xdim * ydim;
	int* a;
	hipMalloc(&a, N * sizeof(int));
	for (int i = 0; i < N; i++) {
		iter[i] = 0;
	}
	hipMemcpy(a, iter, N * sizeof(int), hipMemcpyHostToDevice);

	kernel << < 1200, 800 >> > (a, static_cast<float>(xdim), static_cast<float>(ydim), max_iter);
	hipDeviceSynchronize();

	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		fprintf(stderr, "ERROR: %s\n", hipGetErrorString(error));
		exit(-1);
	}

	hipMemcpy(iter, a, N * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(a);
}